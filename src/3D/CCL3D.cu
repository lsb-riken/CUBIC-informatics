#include "hip/hip_runtime.h"
/*
 * CCL3D.cu
 */

#include <assert.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include "CCL3D.cuh"

#define CCL_BLOCK_SIZE_X 8
#define CCL_BLOCK_SIZE_Y 8
#define CCL_BLOCK_SIZE_Z 8

__device__ int d_isNotDone;

__global__ void initLabels(char* img, int* label, int w, int h, int d) {
    const int x = blockIdx.x * CCL_BLOCK_SIZE_X + threadIdx.x;
    const int y = blockIdx.y * CCL_BLOCK_SIZE_Y + threadIdx.y;
    const int z = blockIdx.z * CCL_BLOCK_SIZE_Z + threadIdx.z;
    const int index = (z*h + y)*w + x;

    if ( x >= w || y >= h || z >= d) return;

    label[index] = index * img[index];
}

__global__ void scanLabels(int* labels, int w, int h, int d) {
    const int x = blockIdx.x * CCL_BLOCK_SIZE_X + threadIdx.x;
    const int y = blockIdx.y * CCL_BLOCK_SIZE_Y + threadIdx.y;
    const int z = blockIdx.z * CCL_BLOCK_SIZE_Z + threadIdx.z;
    const int index = (z*h + y)*w + x;

    if (x >= w || y >= h || z >= d) return;

    const int Z1 = w*h; const int Y1 = w;

    int lcur = labels[index];
    if (lcur) {
      int lmin = index; // MAX
      // 26-neighbors
      int lne, pos;
      for (int Zdif = -Z1; Zdif <= Z1; Zdif += Z1) {
        for (int Ydif = -Y1; Ydif <= Y1; Ydif += Y1) {
          for (int Xdif = -1; Xdif <= 1; Xdif += 1) {
            pos = index + Zdif + Ydif + Xdif;
            lne = (pos >= 0 && pos < w*h*d) ? labels[pos] : 0; // circular boundary
            if (lne && lne < lmin) lmin = lne;
          }
        }
      }
      // need not (Xdif,Ydif,Zdif)=(0,0,0) but no problem

      if (lmin < lcur) {
        int lpa = labels[lcur];
        labels[lpa] = min(lpa, lmin);
        d_isNotDone = 1;
      }
    }
}

__global__ void analyseLabels(int* labels, int w, int h, int d) {
    const int x = blockIdx.x * CCL_BLOCK_SIZE_X + threadIdx.x;
    const int y = blockIdx.y * CCL_BLOCK_SIZE_Y + threadIdx.y;
    const int z = blockIdx.z * CCL_BLOCK_SIZE_Z + threadIdx.z;
    const int index = (z*h + y)*w + x;

    if (x >= w || y >= h || z >= d) return;

    int lcur = labels[index];
    if (lcur) {
      int r = labels[lcur];
      while(r != lcur) {
        lcur = labels[r];
        r = labels[lcur];
      }
      labels[index] = lcur;
    }
}

extern "C" void CCL(char *d_img, int *d_labels, int w, int h, int d) {
  //checkCudaErrors(hipDeviceSetCacheConfig(hipFuncCachePreferShared)); //?

  assert(w % CCL_BLOCK_SIZE_X == 0);
  assert(h % CCL_BLOCK_SIZE_Y == 0);
  assert(d % CCL_BLOCK_SIZE_Z == 0);


  dim3 blocks (ceil(float(w)/CCL_BLOCK_SIZE_X),
               ceil(float(h)/CCL_BLOCK_SIZE_Y),
               ceil(float(d)/CCL_BLOCK_SIZE_Z));
  dim3 threads (CCL_BLOCK_SIZE_X, CCL_BLOCK_SIZE_Y, CCL_BLOCK_SIZE_Z);
  initLabels<<<blocks, threads>>>(d_img, d_labels, w, h, d);
  checkCudaErrors(hipPeekAtLastError());
  checkCudaErrors(hipDeviceSynchronize());

  int isNotDone;
  while(1) {
    isNotDone = 0;
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_isNotDone), &isNotDone, sizeof(int), 0, hipMemcpyHostToDevice));

    //std::cout << "scan " << std::endl;
    scanLabels<<<blocks, threads>>>(d_labels, w, h, d);
    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    checkCudaErrors(hipMemcpyFromSymbol(&isNotDone, HIP_SYMBOL(d_isNotDone), sizeof(int), 0, hipMemcpyDeviceToHost));
    if(isNotDone) {
      //std::cout << "analysis " << std::endl;
      analyseLabels<<<blocks, threads>>>(d_labels, w, h, d);
      checkCudaErrors(hipPeekAtLastError());
      checkCudaErrors(hipDeviceSynchronize());
    }else {
      break;
    }
  }
}
